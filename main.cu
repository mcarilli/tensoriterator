
#include <hip/hip_runtime.h>
#include <iostream>

template <typename T>
void echo_arange(int64_t size) {
  int64_t size_ = size * sizeof(T);

  // set
  T *buf = new T[size];
  for (int64_t i = 0; i < size; i++) {
    buf[i] = T(i);
  }
  T *dev;
  hipMalloc(&dev, size_);
  hipDeviceSynchronize();
  hipMemcpy(dev, buf, size_, hipMemcpyDefault);
  hipDeviceSynchronize();
  delete [] buf;

  // print
  buf = new T[size];
  hipDeviceSynchronize();
  hipMemcpy(buf, dev, size_, hipMemcpyDefault);
  hipDeviceSynchronize();
  for (int64_t i = 0; i < size; i++) {
    std::cout << buf[i] << ", ";
  }
  std::cout << std::endl;
  delete [] buf;
}

int main() {
  echo_arange<float>(30);
}
