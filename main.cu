
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

std::vector<char *> data_ptrs = {
  nullptr, nullptr
};

template <typename T>
T *arange(int64_t size) {
  T *buf = new T[size];
  for (int64_t i = 0; i < size; i++) {
    buf[i] = T(i);
  }
  T *ret;
  int64_t size_ = size * sizeof(T);
  hipMalloc(&ret, size_);
  hipMemcpy(ret, buf, size_, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  delete [] buf;
  // who cares about cudaFree :P LOL
  return ret;
}

template <typename T>
void print(T *data, int64_t size) {
  T *buf = new T[size];
  int64_t size_ = size * sizeof(T);
  hipDeviceSynchronize();
  hipMemcpy(buf, data, size_, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int64_t i = 0; i < size; i++) {
    std::cout << buf[i] << ", ";
  }
  std::cout << std::endl;
  delete [] buf;
}

int main() {
  data_ptrs[0] = (char *)arange<float>(30);
  data_ptrs[1] = (char *)arange<float>(30);
  hipDeviceSynchronize();
  print((float *)data_ptrs[0], 30);
  print((float *)data_ptrs[1], 30);
}
