
#include <hip/hip_runtime.h>
#include <iostream>

template <typename T>
T *arange(int64_t size) {
  T *buf = new T[size];
  for (int64_t i = 0; i < size; i++) {
    buf[i] = T(i);
  }
  T *ret;
  int64_t size_ = size * sizeof(T);
  hipMalloc(&ret, size_);
  hipDeviceSynchronize();
  hipMemcpy(ret, buf, size_, hipMemcpyDefault);
  hipDeviceSynchronize();
  delete [] buf;
  // who cares about cudaFree :P LOL
  return ret;
}

template <typename T>
void print(T *data, int64_t size) {
  T *buf = new T[size];
  int64_t size_ = size * sizeof(T);
  hipDeviceSynchronize();
  hipMemcpy(buf, data, size_, hipMemcpyDefault);
  hipDeviceSynchronize();
  for (int64_t i = 0; i < size; i++) {
    std::cout << buf[i] << ", ";
  }
  std::cout << std::endl;
  delete [] buf;
}

int main() {
  float *a = arange<float>(30);
  hipDeviceSynchronize();
  print(a, 30);
}
